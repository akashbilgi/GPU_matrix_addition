
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    // Calculate global thread ID
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Boundary check
    if (tid < dim) C[tid] = A[tid] + B[tid];

        
    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    /*************************************************************************/
	//dim3 grid(dim,dim);	
    // 
    // 
    int num_thrd = 1 << 10; //1024
    int num_blk = (dim*dim + num_thrd - 1) / num_thrd;


    // Invoke CUDA kernel -----------------------------------------------------
    matAdd<<<num_blk, num_thrd >>>(dim*dim, A, B, C);
    /*************************************************************************/
    //INSERT CODE HERE
	
    /*************************************************************************/

}

